
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

void initWith(float num, float *a, int N)
{
//    int i = threadIdx.x;
    int i = 0;
    for(i = 0; i < N; i++)
        a[i] = num;
}

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < N)
  {
    result[idx] = a[idx] + b[idx];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
  
//  a = (float *)malloc(size);
  hipMallocManaged(&a, size);
  
//  b = (float *)malloc(size);
  hipMallocManaged(&b, size);

//  c = (float *)malloc(size);
  hipMallocManaged(&c, size);

 initWith(3, a, N);
//  initWith<<<1, N>>>(3, a);
  
  initWith(4, b, N);
//  initWith<<<1, N>>>(4, b);
  
  initWith(0, c, N);
//  initWith<<<1, N>>>(0, c);


//  addVectorsInto(c, a, b, N);

  size_t threads_per_block = 256;
  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;
    
  addVectorsInto<<<number_of_blocks,threads_per_block>>>(c, a, b, N);
  checkCuda( hipDeviceSynchronize() );

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
